#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <stdio.h>
#include <iostream>
#include <cmath>

#include "float32.h"

#define IN_DATA_BYTES (IN_SIZE*sizeof(dtype))
#define OUT_DATA_BYTES (OUT_SIZE*sizeof(dtype))

//function to print out error message from cuDNN calls
#define checkCUDNN(exp) \
  { \
    hipdnnStatus_t status = (exp); \
    if(status != HIPDNN_STATUS_SUCCESS) { \
      std::cerr << "Error on line " << __LINE__ << ": " \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE); \
    } \
  }

int main() {
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));

    hipdnnPoolingDescriptor_t pooling_desc;
    //create descriptor handle
    checkCUDNN(hipdnnCreatePoolingDescriptor(&pooling_desc));
    std::cout << "hipdnnCreatePoolingDescriptor is ok...\n";
    //initialize descriptor
    const int poolDims = 2;
    int windowDimA[poolDims] = {2, 2};
    int paddingA[poolDims] = {0, 0};
    int strideA[poolDims] = {2, 2};
    checkCUDNN(hipdnnSetPoolingNdDescriptor(pooling_desc,
                                           HIPDNN_POOLING_MAX,
                                           HIPDNN_PROPAGATE_NAN,
                                           poolDims,
                                           windowDimA,
                                           paddingA,
                                           strideA));

    std::cout << "hipdnnSetPooling2dDescriptor is ok...\n";

    hipdnnTensorDescriptor_t in_desc;
    //create input data tensor descriptor
    checkCUDNN(hipdnnCreateTensorDescriptor(&in_desc));
    std::cout << "hipdnnCreateTensorDescriptor is ok...\n";
    //initialize input data descriptor
    checkCUDNN(hipdnnSetTensor4dDescriptor(in_desc,                  //descriptor handle
                                          HIPDNN_TENSOR_NCHW,        //data format
                                          CUDNN_DTYPE,              //data type (precision)
                                          1,                        //number of images
                                          20,                        //number of channels
                                          24,                       //data height
                                          24));                     //data width
    std::cout << "hipdnnSetTensor4dDescriptor is ok...\n";
    hipdnnTensorDescriptor_t out_desc;
    //create output data tensor descriptor
    checkCUDNN(hipdnnCreateTensorDescriptor(&out_desc));
    std::cout << "hipdnnCreateTensorDescriptor is ok...\n";
    //initialize output data descriptor
    checkCUDNN(hipdnnSetTensor4dDescriptor(out_desc,                 //descriptor handle
                                          HIPDNN_TENSOR_NCHW,        //data format
                                          CUDNN_DTYPE,              //data type (precision)
                                          1,                        //number of images
                                          20,                        //number of channels
                                          12,                        //data height
                                          12));                      //data width

    std::cout << "hipdnnSetTensor4dDescriptor is ok...\n";
    stype alpha = 1.0f;
    stype beta = 0.0f;
    //GPU data pointers
    dtype *in_data, *out_data;
    //allocate arrays on GPU
    hipMalloc(&in_data, IN_DATA_BYTES);
    hipMalloc(&out_data, OUT_DATA_BYTES);
    //copy input data to GPU array
    hipMemcpy(in_data, input, IN_DATA_BYTES, hipMemcpyHostToDevice);
    //initize output data on GPU
    hipMemset(out_data, 0, OUT_DATA_BYTES);

    //Call pooling operator
    checkCUDNN(hipdnnPoolingForward(cudnn,         //cuDNN context handle
                                   pooling_desc,  //pooling descriptor handle
                                   &alpha,        //alpha scaling factor
                                   in_desc,       //input tensor descriptor
                                   in_data,       //input data pointer to GPU memory
                                   &beta,         //beta scaling factor
                                   out_desc,      //output tensor descriptor
                                   out_data));    //output data pointer from GPU memory
    std::cout << "hipdnnPoolingForward is ok...\n";
    //allocate array on CPU for output tensor data
    dtype *result = (dtype *) malloc(OUT_DATA_BYTES);
    //copy output data from GPU
    hipMemcpy(result, out_data, OUT_DATA_BYTES, hipMemcpyDeviceToHost);

    //loop over and check that the forward pass outputs match expected results (exactly)
    int err = 0;
    for (int i = 0; i < OUT_SIZE; i++) {
        if (result[i] != output[i]) {
            std::cout << "Error! Expected " << output[i] << " got " << result[i] << " for idx " << i << std::endl;
            err++;
        }
    }

    std::cout << "Forward finished with " << err << " errors" << std::endl;

    //free CPU arrays
    free(result);

    //free GPU arrays
    hipFree(in_data);
    hipFree(out_data);

    //free cuDNN descriptors
    hipdnnDestroyTensorDescriptor(in_desc);
    hipdnnDestroyTensorDescriptor(out_desc);
    hipdnnDestroyPoolingDescriptor(pooling_desc);
    hipdnnDestroy(cudnn);

    return 0;
}
