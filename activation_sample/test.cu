#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <stdio.h>
#include <iostream>
#include <cmath>

#include "float32.h"

#define IN_DATA_BYTES (IN_SIZE*sizeof(dtype))
#define OUT_DATA_BYTES (OUT_SIZE*sizeof(dtype))

//function to print out error message from cuDNN calls
#define checkCUDNN(exp) \
  { \
    hipdnnStatus_t status = (exp); \
    if(status != HIPDNN_STATUS_SUCCESS) { \
      std::cerr << "Error on line " << __LINE__ << ": " \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE); \
    } \
  }

float getError(float dev, float ref) {
    if (ref > 1.0 || ref < -1.0)
        return (dev - ref) / ref;
    else
        return dev - ref;
}

int main() {
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));

    hipdnnActivationDescriptor_t activDesc;

    checkCUDNN(hipdnnCreateActivationDescriptor(&activDesc));

    checkCUDNN(hipdnnSetActivationDescriptor(activDesc,
                                            HIPDNN_ACTIVATION_RELU,
                                            HIPDNN_PROPAGATE_NAN,
                                            0.0));

    hipdnnTensorDescriptor_t in_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&in_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(in_desc,
                                          HIPDNN_TENSOR_NCHW,
                                          CUDNN_DTYPE,
                                          1, 10,
                                          1,
                                          1));

    hipdnnTensorDescriptor_t out_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&out_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(out_desc,
                                          HIPDNN_TENSOR_NCHW,
                                          CUDNN_DTYPE,
                                          1, 10,
                                          1,
                                          1));

    stype alpha = 1.0f;
    stype beta = 0.0f;
    //GPU data pointers
    dtype *in_data, *out_data;
    //allocate arrays on GPU
    hipMalloc(&in_data, IN_DATA_BYTES);
    hipMalloc(&out_data, OUT_DATA_BYTES);
    //copy input data to GPU array
    hipMemcpy(in_data, input, IN_DATA_BYTES, hipMemcpyHostToDevice);
    //initize output data on GPU
    hipMemset(out_data, 0, OUT_DATA_BYTES);

    checkCUDNN(hipdnnActivationForward(cudnn,
                                      activDesc,
                                      &alpha,
                                      in_desc,
                                      in_data,
                                      &beta,
                                      out_desc,
                                      out_data));

    //allocate array on CPU for output tensor data
    dtype *result = (dtype *) malloc(OUT_DATA_BYTES);
    //copy output data from GPU
    hipMemcpy(result, out_data, OUT_DATA_BYTES, hipMemcpyDeviceToHost);

    //loop over and check that the forward pass outputs match expected results (exactly)
    int err = 0;
    for (int i = 0; i < OUT_SIZE; i++) {
        float diff = getError(result[i], output[i]);
        if (diff < 0) diff = -diff;
        if (diff > 1e-05) {
            std::cout << "Error! Expected " << output[i] << " got " << result[i] << " for idx " << i
                      << std::endl;
            std::cout << "diff " << diff << std::endl;
            err++;
        }
    }

    std::cout << "Forward finished with " << err << " errors" << std::endl;

    //free CPU arrays
    free(result);

    //free GPU arrays
    hipFree(in_data);
    hipFree(out_data);

    //free cuDNN descriptors
    hipdnnDestroyTensorDescriptor(in_desc);
    hipdnnDestroyTensorDescriptor(out_desc);
    hipdnnDestroy(cudnn);

    return 0;
}
