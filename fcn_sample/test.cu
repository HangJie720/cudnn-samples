#include <hipDNN.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include "gemv.h"
#include "float32.h"

#define IN_DATA_BYTES (IN_SIZE*sizeof(dtype))
#define OUT_DATA_BYTES (OUT_SIZE*sizeof(dtype))
#define WEIGHT_DATA_BYTES (WEIGHT_SIZE*sizeof(dtype))
//#define BIAS_DATA_BYTES (BIAS_SIZE*sizeof(dtype))

float getError(float dev, float ref) {
    if (ref > 1.0 || ref < -1.0)
        return (dev - ref) / ref;
    else
        return dev - ref;
}

int main() {
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);

    stype alpha = 1.0f;
    stype beta = 0.0f;

    //GPU data pointers
    dtype *in_data, *out_data;
    dtype *weight_data;

    //allocate arrays on GPU
    hipMalloc(&in_data, IN_DATA_BYTES);
    hipMalloc(&out_data, OUT_DATA_BYTES);
    hipMalloc(&weight_data, WEIGHT_DATA_BYTES);
//    hipMalloc(&bias_data, BIAS_DATA_BYTES);

    //copy input/weight/bias data to GPU array
    hipMemcpy(in_data, input, IN_DATA_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(weight_data, weight, WEIGHT_DATA_BYTES, hipMemcpyHostToDevice);
//    hipMemcpy(bias_data, bias, BIAS_DATA_BYTES, hipMemcpyHostToDevice);

    //initize output data on GPU
    hipMemset(out_data, 0, OUT_DATA_BYTES);

    int dim_x = 4 * 4 * 5;
    int dim_y = 3;
//    hipMemcpy(out_data, bias_data, OUT_DATA_BYTES, hipMemcpyDeviceToDevice);
    //Call fcn operator
    gemv(cublasHandle, dim_x, dim_y, alpha, weight_data, in_data, beta, out_data);

    //allocate array on CPU for output tensor data
    dtype *result = (dtype *) malloc(OUT_DATA_BYTES);
    //copy output data from GPU
    hipMemcpy(result, out_data, OUT_DATA_BYTES, hipMemcpyDeviceToHost);

    //loop over and check that the forward pass outputs match expected results (exactly)
    int err = 0;
    for (int i = 0; i < OUT_SIZE; i++) {
        float diff = getError(result[i], output[i]);
        if (diff < 0) diff = -diff;
        if (diff > 1e-05) {
            std::cout << "Error! Expected " << output[i] << " got " << result[i] << " for idx " << i
                      << std::endl;
            std::cout << "diff " << diff << std::endl;
            err++;
        }
    }

    std::cout << "Forward finished with " << err << " errors" << std::endl;

    //free CPU arrays
    free(result);

    //free GPU arrays
    hipFree(in_data);
    hipFree(out_data);
    hipFree(weight_data);
//    hipFree(bias_data);

    //free cublas descriptors
    hipblasDestroy(cublasHandle);

    return 0;
}
